
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add(int n, float *x, float *y){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = index; i < n; i+=stride) y[i] = x[i] + y[i];
}

int main(void){
    int N = 1<<20; // 1M elemenets
    std::cout << N << std::endl;

    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i = 0; i < N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    ///Run kernel, rounding number of blocks up in case N is not multiple of blocksize
    int blockSize = 256;
    int numBlocks = (N+blockSize-1) / blockSize;
    add <<< numBlocks, 256 >>> (N,x,y);

    //wait for synchro
    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++) maxError = fmax(maxError, fabs(y[i]-3.0f));

    std::cout << "max error: " << maxError << std::endl;

    hipFree(x);
    hipFree(y);
    return 0;
}