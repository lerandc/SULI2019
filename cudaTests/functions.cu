#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "reductions.cuh"
#include <chrono>
#include <random>

__global__
void elementWiseMultiply(int n, float *x, float *y, float *z){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) z[index] = x[index]*y[index];
}

__global__
void reduction(int n, float*x, float*y){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if (idx < n){
        atomicAdd(&y[0], x[idx]);
    }
}

__global__
void reduceMultiply(int n, float*x, float*y, float *z){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if (idx < n){
        atomicAdd(&z[0], x[idx]*y[idx]);
        if(idx == n-1) x[idx] = 10;
    }
}

__global__ void DPC_numerator_reduce(const float* psiIntensity_ds,
									const float* q_coord,
									float* numerator,
									const size_t N){
		int idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx < N){
			atomicAdd(&numerator[0], psiIntensity_ds[idx]*q_coord[idx]);
		}
}

__global__ void DPC_denominator_reduce(const float* psiIntensity_ds,
									float* denominator,
									const size_t N){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		atomicAdd(&denominator[0], psiIntensity_ds[idx]);
	}										   
}

void trial1(const int N){
    for(auto j = 0; j < 100; j++){
         // 1M elemenets
        //std::cout << N << std::endl;
        //std::cout << "stop 0" << std::endl;
        float *x, *y, *z, *q;
        hipMallocManaged(&x, N*sizeof(float));
        hipMallocManaged(&y, N*sizeof(float));
        hipMallocManaged(&q, 1*sizeof(float));
        hipMallocManaged(&z, N*sizeof(float));
        
        //std::cout << "stop 1" << std::endl;
        float *hx = new float[N];
        float *hy = new float[N];
        float *hz = new float[N];
        float *hq = new float[N];
        
        for (int i = 0; i < N; i++){
            hx[i] = 1.0f;
            hz[i] = 0.0f;
            hy[i] = 2.0f;
        }
        hq[0] = 0.0f;
        
        //std::cout << "stop 1" << std::endl;
        hipMemcpy(x,&hx[0],N*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(y,&hy[0],N*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(z,&hz[0],N*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(q,&hq[0],1*sizeof(float),hipMemcpyHostToDevice);
        
        //std::cout << "stop 2" << std::endl;
        ///Run kernel, rounding number of blocks up in case N is not multiple of blocksize
        int blockSize = 256;
        int numBlocks = (N+blockSize-1) / blockSize;
        //elementWiseMultiply <<< numBlocks, 256 >>> (N,x,y,z);
        
        //wait for synchro
        //hipDeviceSynchronize();
        
        reduceMultiply <<<  numBlocks, 256 >>> (N,x,y,q);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(err));
        hipDeviceSynchronize();
        
        //std::cout << "dq: " << q[0] << std::endl;
        //std::cout << "stop 3" << std::endl;
        hipMemcpy(&hx[0],x,N*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(&hy[0],y,N*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(&hz[0],z,N*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(&hq[0],q,1*sizeof(float),hipMemcpyDeviceToHost);
        
        //std::cout << "hq: " << hq[0] << std::endl;
        //std::cout << hq[0] << std::endl;
        //std::cout << hx[N-1] << std::endl;
        
        //std::cout << "stop 4" << std::endl;
        hipFree(x);
        hipFree(y);
        hipFree(q);
        hipFree(z);

        delete [] hx;
        delete [] hy;
        delete [] hz;
        delete [] hq;

    }
}

void trial2(int N){
    for(auto i = 0; i < 100; i++){
        //std::cout << N << std::endl;
        
        float *x, *y, *z, *q;
        hipMallocManaged(&x, N*sizeof(float));
        hipMallocManaged(&y, N*sizeof(float));
        hipMallocManaged(&q, 1*sizeof(float));
        hipMallocManaged(&z, N*sizeof(float));
        
        for (int i = 0; i < N; i++){
            x[i] = 1.0f;
            z[i] = 0.0f;
            y[i] = 2.0f;
        }
        q[0] = 0.0f;
        
        ///Run kernel, rounding number of blocks up in case N is not multiple of blocksize
        int blockSize = 256;
        int numBlocks = (N+blockSize-1) / blockSize;
        reduceMultiply <<< numBlocks, 256 >>> (N,x,y,q);
        
        hipDeviceSynchronize();
        
        hipFree(x);
        hipFree(y);
        hipFree(q);
        hipFree(z);
    }
    
}

int main(void){
    int N = 512;
    double trial1_time = 0.0;
    for(auto j = 0; j < 1; j++){
        auto start = std::chrono::high_resolution_clock::now();
        trial1(N);
        auto stop = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = stop - start;
        std::cout << "Trial 1, test " << j << " elapsed time: " << elapsed.count() << " s\n";
        if(j > 0) trial1_time += elapsed.count(); //ignore startup trial
    }
    trial1_time /= 10;
    //std::cout << "Average for trial 1: " << trial1_time << std::endl;
	//std::cout << "------------------------" << std::endl;
	

	std::random_device rd;
    std::mt19937 e2(rd());
	std::uniform_real_distribution<> dist(0, 1000);

	float qxa[N];
	for(auto i = 0; i < N; i++) qxa[i] = i;
	
	float psi[N];
	for(auto i = 0; i < N; i++) psi[i] = dist(e2)/1000;

	//calculate DPC analogue on CPU side

	//numerator
	float num = 0;
	for(auto i = 0; i < N; i++) num += qxa[i]*psi[i];
	
	//denominator
	float den = 0;
	for(auto i = 0; i < N; i++) den += psi[i];

	//DPC
	std::cout << "CPU DPC_CoM    :" << num/den << std::endl;
	std::cout << "CPU numerator  :" << num << std::endl;
	std::cout << "CPU denominator:" << den << std::endl;
	std::cout << "------------------------" << std::endl;


	//calculate DPC analogue on GPU side

	//allocate variables
	float *qxa_d;
	float *psi_d;
	float *num_d;
	float *den_d;
	hipMallocManaged(&qxa_d,N*sizeof(float));
	hipMallocManaged(&psi_d,N*sizeof(float));
	hipMallocManaged(&num_d,1*sizeof(float));
	hipMallocManaged(&den_d,1*sizeof(float));

	//initialize variables
	float *zero = new float[1];
	zero[0] = 0.0;
	hipMemcpy(qxa_d,&qxa[0],N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(psi_d,&psi[0],N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(num_d,&zero[0],1*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(den_d,&zero[0],1*sizeof(float),hipMemcpyHostToDevice);

	DPC_numerator_reduce <<< 2, 256 >>> (psi_d,qxa_d,num_d,N);
	DPC_denominator_reduce <<< 2, 256 >>> (psi_d,den_d,N);

	float DPC_gpu = num_d[0] / den_d[0];
	std::cout << "GPU DPC_CoM    :" << DPC_gpu << std::endl;
	std::cout << "GPU numerator  :" << num_d[0] << std::endl;
	std::cout << "GPU denominator:" << den_d[0] << std::endl;

	hipFree(qxa_d);
	hipFree(psi_d);
	hipFree(num_d);
	hipFree(den_d);
	free(zero);

    
    return 0;
}



    //for (int i = 0; i < N; i++) std::cout << x[i] << ' '  << y[i] << ' ' << z[i] << std::endl;
    
    /*
    unsigned int bytes = N * sizeof(float);
    
    float *d_idata = NULL;
    float *d_odata = NULL;
    
    float *h_idata = (float *) malloc(bytes);
    
    for (int i=0; i<N; i++){
        h_idata[i] = x[i];
        //std::cout << x[i] << std::endl;
    }
    float *h_odata = (float *) malloc(numBlocks*sizeof(float));
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_odata, h_idata, numBlocks*sizeof(float), hipMemcpyHostToDevice);
    

    blockSize = 32;
    numBlocks = (N+blockSize-1) / blockSize;
    hipMalloc((void **) &d_idata, bytes);
    hipMalloc((void **) &d_odata, numBlocks*sizeof(float));
    
    reduce(N,32,32, d_idata, d_odata);
    //std::cout << d_odata[0] << std::endl;
    float result = 0;
    hipMemcpy(h_odata, d_odata, numBlocks*sizeof(float), hipMemcpyDeviceToHost);
    for (int i=0; i<numBlocks; i++){
        result += h_odata[i];
    }

    std::cout << h_odata[0] << std::endl;
    std::cout << result << std::endl;
    */