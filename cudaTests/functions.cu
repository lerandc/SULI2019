#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "reductions.cuh"
#include <chrono>

__global__
void elementWiseMultiply(int n, float *x, float *y, float *z){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) z[index] = x[index]*y[index];
}

__global__
void reduction(int n, float*x, float*y){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if (idx < n){
        atomicAdd(&y[0], x[idx]);
    }
}

__global__
void reduceMultiply(int n, float*x, float*y, float *z){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if (idx < n){
        atomicAdd(&z[0], x[idx]*y[idx]);
    }
}


void trial1(int N){
    for(auto j = 0; j < 100; j++){
         // 1M elemenets
        //std::cout << N << std::endl;
        //std::cout << "stop 0" << std::endl;
        float *x, *y, *z, *q;
        hipMallocManaged(&x, N*sizeof(float));
        hipMallocManaged(&y, N*sizeof(float));
        hipMallocManaged(&q, 1*sizeof(float));
        hipMallocManaged(&z, N*sizeof(float));
        
        //std::cout << "stop 1" << std::endl;
        float hx[N], hy[N], hz[N], hq[1];
        
        for (int i = 0; i < N; i++){
            hx[i] = 1.0f;
            hz[i] = 0.0f;
            hy[i] = 2.0f;
        }
        hq[0] = 0.0f;
        
        //std::cout << "stop 1" << std::endl;
        hipMemcpy(x,&hx,N*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(y,&hy,N*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(z,&hz,N*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(q,&hq,1*sizeof(float),hipMemcpyHostToDevice);
        
        //std::cout << "stop 2" << std::endl;
        ///Run kernel, rounding number of blocks up in case N is not multiple of blocksize
        int blockSize = 256;
        int numBlocks = (N+blockSize-1) / blockSize;
        elementWiseMultiply <<< numBlocks, 256 >>> (N,x,y,z);
        
        //wait for synchro
        hipDeviceSynchronize();
        
        reduction <<<  numBlocks, 256 >>> (N,z,q);
        
        hipDeviceSynchronize();
        
        //std::cout << "stop 3" << std::endl;
        hipMemcpy(hx,x,N*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(hy,y,N*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(hz,z,N*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(hq,q,1*sizeof(float),hipMemcpyDeviceToHost);
        
        std::cout << hz[0] << std::endl;
        
        //std::cout << "stop 4" << std::endl;
        hipFree(x);
        hipFree(y);
        hipFree(q);
        hipFree(z);
    }
}

void trial2(int N){
    for(auto i = 0; i < 100; i++){
        //std::cout << N << std::endl;
        
        float *x, *y, *z, *q;
        hipMallocManaged(&x, N*sizeof(float));
        hipMallocManaged(&y, N*sizeof(float));
        hipMallocManaged(&q, 1*sizeof(float));
        hipMallocManaged(&z, N*sizeof(float));
        
        for (int i = 0; i < N; i++){
            x[i] = 1.0f;
            z[i] = 0.0f;
            y[i] = 2.0f;
        }
        q[0] = 0.0f;
        
        ///Run kernel, rounding number of blocks up in case N is not multiple of blocksize
        int blockSize = 256;
        int numBlocks = (N+blockSize-1) / blockSize;
        reduceMultiply <<< numBlocks, 256 >>> (N,x,y,q);
        
        hipDeviceSynchronize();
        
        hipFree(x);
        hipFree(y);
        hipFree(q);
        hipFree(z);
    }
    
}

int main(void){
    int N = 512;
    double trial1_time = 0.0;
    for(auto j = 0; j < 11; j++){
        auto start = std::chrono::high_resolution_clock::now();
        trial1(N);
        auto stop = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = stop - start;
        std::cout << "Trial 1, test " << j << " elapsed time: " << elapsed.count() << " s\n";
        if(j > 0) trial1_time += elapsed.count(); //ignore startup trial
    }
    trial1_time /= 10;
    std::cout << "Average for trial 1: " << trial1_time << std::endl;
    std::cout << "------------------------" << std::endl;
    
    return 0;
}



    //for (int i = 0; i < N; i++) std::cout << x[i] << ' '  << y[i] << ' ' << z[i] << std::endl;
    
    /*
    unsigned int bytes = N * sizeof(float);
    
    float *d_idata = NULL;
    float *d_odata = NULL;
    
    float *h_idata = (float *) malloc(bytes);
    
    for (int i=0; i<N; i++){
        h_idata[i] = x[i];
        //std::cout << x[i] << std::endl;
    }
    float *h_odata = (float *) malloc(numBlocks*sizeof(float));
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_odata, h_idata, numBlocks*sizeof(float), hipMemcpyHostToDevice);
    

    blockSize = 32;
    numBlocks = (N+blockSize-1) / blockSize;
    hipMalloc((void **) &d_idata, bytes);
    hipMalloc((void **) &d_odata, numBlocks*sizeof(float));
    
    reduce(N,32,32, d_idata, d_odata);
    //std::cout << d_odata[0] << std::endl;
    float result = 0;
    hipMemcpy(h_odata, d_odata, numBlocks*sizeof(float), hipMemcpyDeviceToHost);
    for (int i=0; i<numBlocks; i++){
        result += h_odata[i];
    }

    std::cout << h_odata[0] << std::endl;
    std::cout << result << std::endl;
    */